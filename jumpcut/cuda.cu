#include "hip/hip_runtime.h"
// includes CUDA Runtime
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>

#include <algorithm>
#include <limits>
#include "BITMAP3.h"
#include "hip/hip_runtime.h"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
	printf("Error at %s:%d\n", __FILE__, __LINE__); \
	return EXIT_FAILURE; }} while (0)

#define MAX_MEXTURE_WIDTH 65536

int BLOCKDIM = 8;
texture<int1, 2, hipReadModeElementType> texImageA, texImageB;
texture<float1, 1, hipReadModeElementType> sourceEdgeTex, targetEdgeTex;
hipArray *a_Src, *b_Src;

__device__ float sqrt7(float x)
{
	unsigned int i = *(unsigned int*)&x;
	// adjust bias
	i += 127 << 23;
	// approximation of square root
	i >>= 1;
	return *(float*)&i;
}

__device__ float sqrt3(const float x)
{
	union
	{
		int i;
		float x;
	} u;

	u.x = x;
	u.i = (1 << 29) + (u.i >> 1) - (1 << 22);
	return u.x;
}

int iDivUp(int a, int b)
{
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__device__ void simpleImproveGuess(int ax, int ay, int xp, int yp, int &xbest, int &ybest, int &dbest, int patch_w)
{
	int ans = 0;
	for (int dy = 0; dy < patch_w; dy++) {
		for (int dx = 0; dx < patch_w; dx++) {
			int1 ac1 = tex2D(texImageA, ax + dx, ay + dy);
			int1 bc1 = tex2D(texImageB, xp + dx, yp + dy);
			int ac = ac1.x, bc = bc1.x;
			int dr = (ac & 255) - (bc & 255);
			int dg = ((ac >> 8) & 255) - ((bc >> 8) & 255);
			int db = (ac >> 16) - (bc >> 16);
			ans += dr*dr + dg*dg + db*db;
		}
	}

	if (ans < dbest) {
		dbest = ans;
		xbest = xp;
		ybest = yp;
	}
}

__device__ void improve_guess(int ax, int ay, int xp, int yp, int &xbest, int &ybest, int &dbest, int diagonalDis, bool leapPropagate, int patch_w)
{
	float alpha = 10, beta = 3;
	if (leapPropagate)
		beta = 1.5;
	double ansColor = 0, ansDis = 0, ans = 0;
	double dxx = ax - xp, dyy = ay - yp;
	double dis = sqrt7(dxx*dxx + dyy*dyy) / diagonalDis*100.0;
	ansDis = dis*patch_w*patch_w;

	for (int dy = 0; dy < patch_w; dy++) {
		for (int dx = 0; dx < patch_w; dx++) {
			int1 ac1 = tex2D(texImageA, ax + dx, ay + dy);
			int1 bc1 = tex2D(texImageB, xp + dx, yp + dy);
			int ac = ac1.x, bc = bc1.x;
			int dr = (ac & 255) - (bc & 255);
			int dg = ((ac >> 8) & 255) - ((bc >> 8) & 255);
			int db = (ac >> 16) - (bc >> 16);
			ansColor += sqrt7((double)(dr*dr + dg*dg + db*db));
		}
	}
	ansColor = ansColor / 255.0*100.0;
	ans = ansColor*alpha + ansDis*beta;

	if (ans < dbest) {
		dbest = ans;
		xbest = xp;
		ybest = yp;
	}
}

__device__ int RNG(int idx, int idy)
{
	unsigned int m_w = idx;
	unsigned int m_z = idy;

	m_z = 36969 * (m_z & 65535) + (m_z >> 16);
	m_w = 18000 * (m_w & 65535) + (m_w >> 16);

	return abs((int)((m_z << 16) + m_w));
}

__device__ unsigned int g_seed = 0;
__device__ inline int fastrand()
{
	g_seed = (214013 * g_seed + 2531011);
	return (g_seed >> 16) & 0x7FFF;
}

__device__ static unsigned int z1 = 12345, z2 = 12345, z3 = 12345, z4 = 12345;
__device__ unsigned int lfsr113_Bits(void)
{
	unsigned int b;
	b = ((z1 << 6) ^ z1) >> 13;
	z1 = ((z1 & 4294967294U) << 18) ^ b;
	b = ((z2 << 2) ^ z2) >> 27;
	z2 = ((z2 & 4294967288U) << 2) ^ b;
	b = ((z3 << 13) ^ z3) >> 21;
	z3 = ((z3 & 4294967280U) << 7) ^ b;
	b = ((z4 << 3) ^ z4) >> 12;
	z4 = ((z4 & 4294967168U) << 13) ^ b;
	return (z1 ^ z2 ^ z3 ^ z4);
}

__global__ void setup_kernel(hiprandState *state, int aew, int aeh)
{
	int  idx = blockIdx.x * blockDim.x + threadIdx.x;
	int  idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx < aew && idy < aeh)
	{
		int index = idx + idy * aew;
		hiprand_init(index, 0, 0, &state[index]);
	}
}

__global__ void kernel_propagation2(hiprandState *state, int *ann, int *annd, int height, int width, int min_wh, int diagonalDis, bool leapPropagate, int patch_w, int patch_iter)
{
	int  idx = blockIdx.x * blockDim.x + threadIdx.x;
	int  idy = blockIdx.y * blockDim.y + threadIdx.y;

	int aew = width - patch_w + 1, aeh = height - patch_w + 1;
	for (int iter = 0; iter < patch_iter; iter++)
	{
		int temp_iter = iter;
		if (iter>patch_iter / 2)
			temp_iter = patch_iter - iter;
		int change = pow((double)2.0, (double)temp_iter);
		if (idx < aew && idy < aeh)
		{
			//printf("a\n");
			int xchange = change, ychange = change;

			int index = idx + idy * aew;
			hiprandState localState = state[index];

			//hiprandState localState;
			//hiprand_init(index, 0, 0, &localState);

			/* Current (best) guess. */
			int v = ann[idy * width + idx];
			int xbest = INT_TO_X(v), ybest = INT_TO_Y(v);
			int dbest = annd[idy * width + idx];

			/* Propagation: Improve current guess by trying instead correspondences from left and above (below and right on odd iterations). */
			if ((unsigned)(idx - xchange) < (unsigned)aew) {
				int vp = ann[idy * width + idx - xchange];
				int xp = INT_TO_X(vp) + xchange, yp = INT_TO_Y(vp);
				if ((unsigned)xp < (unsigned)aew) {
					improve_guess(idx, idy, xp, yp, xbest, ybest, dbest, diagonalDis, leapPropagate, patch_w);
				}
			}

			if ((unsigned)(idx + xchange) < (unsigned)aew) {
				int vp = ann[idy * width + idx + xchange];
				int xp = INT_TO_X(vp) - xchange, yp = INT_TO_Y(vp);
				if ((unsigned)xp < (unsigned)aew) {
					improve_guess(idx, idy, xp, yp, xbest, ybest, dbest, diagonalDis, leapPropagate, patch_w);
				}
			}

			if ((unsigned)(idy - ychange) < (unsigned)aeh) {
				int vp = ann[(idy - ychange) * width + idx];
				int xp = INT_TO_X(vp), yp = INT_TO_Y(vp) + ychange;
				if ((unsigned)yp < (unsigned)aeh) {
					improve_guess(idx, idy, xp, yp, xbest, ybest, dbest, diagonalDis, leapPropagate, patch_w);
				}
			}

			if ((unsigned)(idy + ychange) < (unsigned)aeh) {
				int vp = ann[(idy + ychange) * width + idx];
				int xp = INT_TO_X(vp), yp = INT_TO_Y(vp) - ychange;
				if ((unsigned)yp < (unsigned)aeh) {
					improve_guess(idx, idy, xp, yp, xbest, ybest, dbest, diagonalDis, leapPropagate, patch_w);
				}
			}

			/* Random search: Improve current guess by searching in boxes of exponentially decreasing size around the current best guess. */
			int rs_start = min_wh;
			if (rs_start > MAX(width, height)) { rs_start = MAX(width, height); }
			for (int mag = rs_start; mag >= 1; mag /= 2)
			{
				/* Sampling window */
				int xmin = MAX(xbest - mag, 0), xmax = MIN(xbest + mag + 1, aew);
				int ymin = MAX(ybest - mag, 0), ymax = MIN(ybest + mag + 1, aeh);
				//int xp = xmin + fastrand() % (xmax - xmin);
				//int yp = ymin + fastrand() % (ymax - ymin);

				//int xp = xmin + lfsr113_Bits() % (xmax - xmin);
				//int yp = ymin + lfsr113_Bits() % (ymax - ymin);

				//int xp = xmin + RNG(idx, idy) % (xmax - xmin);
				//int yp = ymin + RNG(idx, idy) % (ymax - ymin);

				int xp = xmin + hiprand(&localState) % (xmax - xmin);
				int yp = ymin + hiprand(&localState) % (ymax - ymin);
				improve_guess(idx, idy, xp, yp, xbest, ybest, dbest, diagonalDis, leapPropagate, patch_w);
			}

			ann[idy * width + idx] = XY_TO_INT(xbest, ybest);
			annd[idy *width + idx] = dbest;
			state[index] = localState;
		}
	}
}

__global__ void kernel_propagation(hiprandState *state, int *ann, int *annd, int height, int width, int min_wh, int diagonalDis, bool leapPropagate, int patch_w, int change)
{
	int  idx = blockIdx.x * blockDim.x + threadIdx.x;
	int  idy = blockIdx.y * blockDim.y + threadIdx.y;

	int aew = width - patch_w + 1, aeh = height - patch_w + 1;
	if (idx < aew && idy < aeh)
	{
		//printf("a\n");
		int xchange = change, ychange = change;

		int index = idx + idy * aew;
		hiprandState localState = state[index];

		//hiprandState localState;
		//hiprand_init(index, 0, 0, &localState);

		/* Current (best) guess. */
		int v = ann[idy * width + idx];
		int xbest = INT_TO_X(v), ybest = INT_TO_Y(v);
		int dbest = annd[idy * width + idx];

		/* Propagation: Improve current guess by trying instead correspondences from left and above (below and right on odd iterations). */
		if ((unsigned)(idx - xchange) < (unsigned)aew) {
			int vp = ann[idy * width + idx - xchange];
			int xp = INT_TO_X(vp) + xchange, yp = INT_TO_Y(vp);
			if ((unsigned)xp < (unsigned)aew) {
				improve_guess(idx, idy, xp, yp, xbest, ybest, dbest, diagonalDis, leapPropagate, patch_w);
			}
		}

		if ((unsigned)(idx + xchange) < (unsigned)aew) {
			int vp = ann[idy * width + idx + xchange];
			int xp = INT_TO_X(vp) - xchange, yp = INT_TO_Y(vp);
			if ((unsigned)xp < (unsigned)aew) {
				improve_guess(idx, idy, xp, yp, xbest, ybest, dbest, diagonalDis, leapPropagate, patch_w);
			}
		}

		if ((unsigned)(idy - ychange) < (unsigned)aeh) {
			int vp = ann[(idy - ychange) * width + idx];
			int xp = INT_TO_X(vp), yp = INT_TO_Y(vp) + ychange;
			if ((unsigned)yp < (unsigned)aeh) {
				improve_guess(idx, idy, xp, yp, xbest, ybest, dbest, diagonalDis, leapPropagate, patch_w);
			}
		}

		if ((unsigned)(idy + ychange) < (unsigned)aeh) {
			int vp = ann[(idy + ychange) * width + idx];
			int xp = INT_TO_X(vp), yp = INT_TO_Y(vp) - ychange;
			if ((unsigned)yp < (unsigned)aeh) {
				improve_guess(idx, idy, xp, yp, xbest, ybest, dbest, diagonalDis, leapPropagate, patch_w);
			}
		}

		/* Random search: Improve current guess by searching in boxes of exponentially decreasing size around the current best guess. */
		int rs_start = min_wh;
		if (rs_start > MAX(width, height)) { rs_start = MAX(width, height); }
		for (int mag = rs_start; mag >= 1; mag /= 2)
		{
			/* Sampling window */
			int xmin = MAX(xbest - mag, 0), xmax = MIN(xbest + mag + 1, aew);
			int ymin = MAX(ybest - mag, 0), ymax = MIN(ybest + mag + 1, aeh);

			int xp = xmin + hiprand(&localState) % (xmax - xmin);
			int yp = ymin + hiprand(&localState) % (ymax - ymin);
			improve_guess(idx, idy, xp, yp, xbest, ybest, dbest, diagonalDis, leapPropagate, patch_w);
		}

		ann[idy * width + idx] = XY_TO_INT(xbest, ybest);
		annd[idy *width + idx] = dbest;
		state[index] = localState;
	}
}

__global__ void simpleKernelProp(hiprandState *state, int *ann, int *annd, int height, int width, int patch_w, int change)
{
	int  idx = blockIdx.x * blockDim.x + threadIdx.x;
	int  idy = blockIdx.y * blockDim.y + threadIdx.y;

	int aew = width - patch_w + 1, aeh = height - patch_w + 1;
	if (idx < aew && idy < aeh)
	{
		//printf("a\n");
		int xchange = change, ychange = change;

		int index = idx + idy * aew;
		hiprandState localState = state[index];

		//hiprandState localState;
		//hiprand_init(index, 0, 0, &localState);

		/* Current (best) guess. */
		int v = ann[idy * width + idx];
		int xbest = INT_TO_X(v), ybest = INT_TO_Y(v);
		int dbest = annd[idy * width + idx];

		/* Propagation: Improve current guess by trying instead correspondences from left and above (below and right on odd iterations). */
		if ((unsigned)(idx - xchange) < (unsigned)aew) {
			int vp = ann[idy * width + idx - xchange];
			int xp = INT_TO_X(vp) + xchange, yp = INT_TO_Y(vp);
			if ((unsigned)xp < (unsigned)aew) {
				simpleImproveGuess(idx, idy, xp, yp, xbest, ybest, dbest, patch_w);
			}
		}

		if ((unsigned)(idx + xchange) < (unsigned)aew) {
			int vp = ann[idy * width + idx + xchange];
			int xp = INT_TO_X(vp) - xchange, yp = INT_TO_Y(vp);
			if ((unsigned)xp < (unsigned)aew) {
				simpleImproveGuess(idx, idy, xp, yp, xbest, ybest, dbest, patch_w);
			}
		}

		if ((unsigned)(idy - ychange) < (unsigned)aeh) {
			int vp = ann[(idy - ychange) * width + idx];
			int xp = INT_TO_X(vp), yp = INT_TO_Y(vp) + ychange;
			if ((unsigned)yp < (unsigned)aeh) {
				simpleImproveGuess(idx, idy, xp, yp, xbest, ybest, dbest, patch_w);
			}
		}

		if ((unsigned)(idy + ychange) < (unsigned)aeh) {
			int vp = ann[(idy + ychange) * width + idx];
			int xp = INT_TO_X(vp), yp = INT_TO_Y(vp) - ychange;
			if ((unsigned)yp < (unsigned)aeh) {
				simpleImproveGuess(idx, idy, xp, yp, xbest, ybest, dbest, patch_w);
			}
		}

		/* Random search: Improve current guess by searching in boxes of exponentially decreasing size around the current best guess. */
		int rs_start = INT_MAX;
		if (rs_start > MAX(width, height)) { rs_start = MAX(width, height); }
		for (int mag = rs_start; mag >= 1; mag /= 2)
		{
			/* Sampling window */
			int xmin = MAX(xbest - mag, 0), xmax = MIN(xbest + mag + 1, aew);
			int ymin = MAX(ybest - mag, 0), ymax = MIN(ybest + mag + 1, aeh);

			int xp = xmin + hiprand(&localState) % (xmax - xmin);
			int yp = ymin + hiprand(&localState) % (ymax - ymin);
			simpleImproveGuess(idx, idy, xp, yp, xbest, ybest, dbest, patch_w);
		}

		ann[idy * width + idx] = XY_TO_INT(xbest, ybest);
		annd[idy *width + idx] = dbest;
		state[index] = localState;
	}
}

extern "C"
void propagation(BITMAP3 *a, BITMAP3 *b, BITMAP3 *&ann, BITMAP3 *&annd, int min_wh, int diagonalDis, bool leapPropagate)
{
	FILE *file = fopen(".\propagation.txt", "w");
	int width = a->w, height = a->h;
	int aew = width - global_patch_w + 1, aeh = height - global_patch_w + 1;
	int *gpu_ann, *gpu_annd;	

	int sz = sizeof(int)*width*height;
	checkCudaErrors(hipMalloc((void**)&gpu_ann, sz));
	checkCudaErrors(hipMalloc((void**)&gpu_annd, sz));
	checkCudaErrors(hipMemcpy(gpu_ann, ann->data, sz, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(gpu_annd, annd->data, sz, hipMemcpyHostToDevice));

	checkCudaErrors(hipMallocArray(&a_Src, &texImageA.channelDesc, width, height));
	checkCudaErrors(hipMallocArray(&b_Src, &texImageB.channelDesc, width, height));
	checkCudaErrors(hipMemcpyToArray(a_Src, 0, 0, a->data, sz, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpyToArray(b_Src, 0, 0, b->data, sz, hipMemcpyHostToDevice));

	texImageA.normalized = false;
	texImageA.addressMode[0] = hipAddressModeClamp;
	texImageA.addressMode[1] = hipAddressModeClamp;
	texImageA.filterMode = hipFilterModePoint;
	texImageB.normalized = false;
	texImageB.addressMode[0] = hipAddressModeClamp;
	texImageB.addressMode[1] = hipAddressModeClamp;
	texImageB.filterMode = hipFilterModePoint;

	checkCudaErrors(hipBindTextureToArray(texImageA, a_Src));
	checkCudaErrors(hipBindTextureToArray(texImageB, b_Src));

	dim3 gridDim(ceil((float)aew / BLOCKDIM), ceil((float)aeh / BLOCKDIM));
	dim3 blockDim(BLOCKDIM, BLOCKDIM);

	hiprandState *state;
	checkCudaErrors(hipMalloc((void**)&state, aew*aeh*sizeof(hiprandState)));

	setup_kernel << <gridDim, blockDim >> >(state, aew, aeh);
	hipDeviceSynchronize();

	for (int iter = 0; iter < global_pm_iters; iter++)
	{
		int temp_iter = iter;
		int change = pow((double)2.0, (double)temp_iter);
		if (iter == global_pm_iters - 2)
			change = 2;
		if (iter == global_pm_iters - 1)
			change = 1;
		hipDeviceSynchronize();
		
		kernel_propagation << <gridDim, blockDim >> >(state, gpu_ann, gpu_annd, height, width, min_wh, diagonalDis, leapPropagate, global_patch_w, change);
		hipDeviceSynchronize();
	}
	
	checkCudaErrors(hipMemcpy(ann->data, gpu_ann, sz, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(annd->data, gpu_annd, sz, hipMemcpyDeviceToHost));

	//for (int y = 0; y < aeh; ++y)
	//{
	//	for (int x = 0; x < aew; ++x)
	//	{
	//		int v = ann->data[y*width + x];
	//		int xx = INT_TO_X(v), yy = INT_TO_Y(v);
	//		if (xx >= aew || yy >= aeh || xx < 0 || yy < 0)
	//		{
	//			fprintf(file, "original pos: %d, %d. new pos: %d, %d.\n", x, y, xx, yy);
	//		}
	//	}
	//}

	checkCudaErrors(hipFree(state));
	checkCudaErrors(hipFree(gpu_ann));
	checkCudaErrors(hipFree(gpu_annd));
	checkCudaErrors(hipUnbindTexture(texImageA));
	checkCudaErrors(hipUnbindTexture(texImageB));
	checkCudaErrors(hipFreeArray(a_Src));
	checkCudaErrors(hipFreeArray(b_Src));
	fclose(file);
}

__device__ void quicksort(float *dists, int *idx, int first, int last)
{
	int pivot, j, i, temp_idx;
	float temp_dist;

	if (first < last)
	{
		pivot = first;
		i = first;
		j = last;

		while (i < j){
			while (dists[i] <= dists[pivot] && i<last)
				i++;
			while (dists[j]>dists[pivot])
				j--;
			if (i < j){
				temp_dist = dists[i];
				dists[i] = dists[j];
				dists[j] = temp_dist;

				temp_idx = idx[i];
				idx[i] = idx[j];
				idx[j] = temp_idx;
			}
		}

		temp_dist = dists[pivot];
		dists[pivot] = dists[j];
		dists[j] = temp_dist;

		temp_idx = idx[pivot];
		idx[pivot] = idx[j];
		idx[j] = temp_idx;

		quicksort(dists, idx, first, j - 1);
		quicksort(dists, idx, j + 1, last);
	}
}

void quicksort_host(float *dists, int *idx, int first, int last)
{
	int pivot, j, i, temp_idx;
	float temp_dist;

	if (first < last)
	{
		pivot = first;
		i = first;
		j = last;

		while (i < j){
			while (dists[i] <= dists[pivot] && i<last)
				i++;
			while (dists[j]>dists[pivot])
				j--;
			if (i < j){
				temp_dist = dists[i];
				dists[i] = dists[j];
				dists[j] = temp_dist;

				temp_idx = idx[i];
				idx[i] = idx[j];
				idx[j] = temp_idx;
			}
		}

		temp_dist = dists[pivot];
		dists[pivot] = dists[j];
		dists[j] = temp_dist;

		temp_idx = idx[pivot];
		idx[pivot] = idx[j];
		idx[j] = temp_idx;

		quicksort_host(dists, idx, first, j - 1);
		quicksort_host(dists, idx, j + 1, last);
	}
}

__global__ void computeAllDists(int *&idxs, float *&dists, int sourceHeight, int targetHeight, int elementSize, int searchSize)
{
	unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if (xIndex < targetHeight)
	{
		//float *vecDist = (float *)malloc(sourceHeight*sizeof(float));
		//int *vecIdx = (int *)malloc(sourceHeight*sizeof(int));
		for (int y = 0; y < sourceHeight; y++)
		{
			float sum = 0;
			for (int x = 0; x < elementSize; x++)
			{
				sum += fabs(tex1Dfetch(targetEdgeTex, xIndex*elementSize + x).x - tex1Dfetch(sourceEdgeTex, y*elementSize + x).x);
			}

			//vecDist[y] = sum;
			//vecIdx[y] = y;
		}

		//quicksort(vecDist, vecIdx, 0, sourceHeight - 1);
		//memcpy(dists + sizeof(float)*searchSize*xIndex, vecDist, sizeof(float)*searchSize);
		//memcpy(idxs + sizeof(int)*searchSize*xIndex, vecIdx, sizeof(int)*searchSize);
		//free(vecDist); vecDist = NULL;
		//free(vecIdx); vecIdx = NULL;
	}
}

void computeAllDists_host(float *source, float *target, int *&idxs, float *&dists, int sourceHeight, int targetHeight, int elementSize, int searchSize)
{
	for (int xIndex = 0; xIndex < targetHeight; xIndex++)
	{
		float *vecDist = (float *)malloc(sourceHeight*sizeof(float));
		int *vecIdx = (int *)malloc(sourceHeight*sizeof(int));
		for (int y = 0; y < sourceHeight; y++)
		{
			float sum = 0;
			for (int x = 0; x < elementSize; x++)
			{
				sum += fabs(target[xIndex*elementSize + x] - source[y*elementSize + x]);
			}

			vecDist[y] = sum;
			vecIdx[y] = y;
		}

		quicksort_host(vecDist, vecIdx, 0, sourceHeight - 1);
		memcpy(dists + sizeof(float)*searchSize*xIndex, vecDist, sizeof(float)*searchSize);
		memcpy(idxs + sizeof(int)*searchSize*xIndex, vecIdx, sizeof(int)*searchSize);
		free(vecDist);
		free(vecIdx);
	}
}

extern "C" void knnSearch(float *sourceSet, float *targetSet, int *m_indices, float *m_dists, int sourceHeight, int targetHeight, int elementNum, int searchSize)
{
	float *source, *target, *dists;
	int *idxs;

	size_t sourceSize = sourceHeight * elementNum * sizeof(float);
	size_t targetSize = targetHeight * elementNum * sizeof(float);
	size_t idxSize = targetHeight * searchSize * sizeof(int);
	size_t distSize = targetHeight * searchSize * sizeof(float);

	checkCudaErrors(hipMalloc((void **)&source, sourceSize));
	checkCudaErrors(hipMalloc((void **)&target, targetSize));
	checkCudaErrors(hipMalloc((void **)&idxs, idxSize));
	checkCudaErrors(hipMalloc((void **)&dists, distSize));

	checkCudaErrors(hipMemcpy(idxs, m_indices, idxSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dists, m_dists, distSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(source, sourceSet, sourceSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(target, targetSet, targetSize, hipMemcpyHostToDevice));

	sourceEdgeTex.normalized = false;
	sourceEdgeTex.addressMode[0] = hipAddressModeClamp;
	sourceEdgeTex.filterMode = hipFilterModePoint;
	targetEdgeTex.normalized = false;
	targetEdgeTex.addressMode[0] = hipAddressModeClamp;
	targetEdgeTex.filterMode = hipFilterModePoint;

	checkCudaErrors(hipBindTexture(0, sourceEdgeTex, source, sourceEdgeTex.channelDesc));
	checkCudaErrors(hipBindTexture(0, targetEdgeTex, target, targetEdgeTex.channelDesc));

	dim3 blockDim(256);
	dim3 gridDim(iDivUp(targetHeight, blockDim.x));

	computeAllDists << <gridDim, blockDim >> >(idxs, dists, sourceHeight, targetHeight, elementNum, searchSize);

	//checkCudaErrors(hipMemcpy(m_indices, idxs, idxSize, hipMemcpyDeviceToHost));
	//checkCudaErrors(hipMemcpy(m_dists, dists, distSize, hipMemcpyDeviceToHost));
	//checkCudaErrors(hipMemcpy(sourceSet, source, sourceSize, hipMemcpyDeviceToHost));
	//checkCudaErrors(hipMemcpy(targetSet, target, targetSize, hipMemcpyDeviceToHost));
	//computeAllDists_host(sourceSet, targetSet, m_indices, m_dists, sourceHeight, targetHeight, elementNum, searchSize);

	checkCudaErrors(hipPeekAtLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpy(m_indices, idxs, idxSize, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(m_dists, dists, distSize, hipMemcpyDeviceToHost));

	checkCudaErrors(hipUnbindTexture(sourceEdgeTex));
	checkCudaErrors(hipUnbindTexture(targetEdgeTex));
	checkCudaErrors(hipFree(idxs));
	checkCudaErrors(hipFree(dists));
	checkCudaErrors(hipFree(source));
	checkCudaErrors(hipFree(target));
}